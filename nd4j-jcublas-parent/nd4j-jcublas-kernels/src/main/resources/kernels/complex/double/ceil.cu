extern "C"

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>
__global__ void ceil_double(int n,int idx,double *dy,int incy,double *result) {
             for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                            if(i >= idx && i % incy == 0)
                                result[i] =  ceil(dy[i]);
              }
}